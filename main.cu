
#include <hip/hip_runtime.h>
#define N 8

__global__ void vec_add(float *A, float *B, float *C)
{
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
}

int main()
{
	float A[N] = {1, 2, 3, 4, 5, 6, 7, 8};
	float B[N] = {8, 7, 6, 5, 4, 3, 2, 1};
	float C[N] = {};

	vec_add<<<1, N>>>(A, B, C);

	printf("hello world");

	return 0;
}
